#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#ifdef GUI
#include <GL/glut.h>
#include <GL/gl.h>
#include <GL/glu.h>
#endif

#include "./headers/physics.h"
#include "./headers/logger.h"


int block_size = 512;


int n_body;
int n_iteration;


__global__ void update_position(double *x, double *y, double *vx, double *vy, int n) {
    //TODO: update position 
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        x[i] = x[i] + vx[i] * dt;
        y[i] = y[i] + vy[i] * dt;
    }
}

__global__ void update_velocity(double *m, double *x, double *y, double *vx, double *vy, int n) {
    //TODO: calculate force and acceleration, update velocity
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {  
        double acceleration = 0;
        double acceleration_x = 0;
        double acceleration_y = 0;
        // double origin_vx, origin_vy;
        double x_proj, y_proj, xy_distance_pow;
        if (x[i] > bound_x || x[i] < 0. || y[i] > bound_y || y[i] < 0.)
        {
            vx[i] = -vx[i];
            vy[i] = -vy[i];
            // printf("OVER HERE #1\n");
        }
        // printf("OVER HERE #1\n");
        for (int j = 0; j < n; j++)
        {
            if (i != j)
            {
                xy_distance_pow = pow(x[i] - x[j], 2.) + pow(y[i] - y[j], 2.);
                // xy_distance_pow = (x[i] - x[j]) * (x[i] - x[j]) + (y[i] - y[j]) * (y[i] - y[j]);
                // printf("ans111:%f,\n", pow(x[i] - x[j], 2.));
                // printf("ans2:%f,\n", pow(y[i] - y[j], 2.));
                // printf("xi:%f, xj:%f, yi:%f, yj:%f\n", x[i], x[j], y[i], y[j]);
                // printf("xy_dis:%f,\n", xy_distance_pow);
                if (xy_distance_pow < 1000 * radius2)
                {
                    vx[i] = -vx[i];
                    vy[i] = -vy[i];
                    acceleration_x = 0;
                    acceleration_y = 0;
                    break;
                }
                else
                {
                    x_proj = pow(pow(x[i] - x[j], 2.) / xy_distance_pow, 0.5);
                    y_proj = pow(pow(y[i] - y[j], 2.) / xy_distance_pow, 0.5);
                    acceleration = gravity_const * m[j] / (xy_distance_pow + err);
                    // printf("acc:%f, xP:%f, yP:%f\n", acceleration, x_proj, y_proj);
                    if (x[i] < x[j])
                    {
                        acceleration_x = acceleration_x + acceleration * x_proj;
                    } else {
                        acceleration_x = acceleration_x - acceleration * x_proj;
                    }
                    if (y[i] < y[j])
                    {
                        acceleration_y = acceleration_y + acceleration * y_proj;
                    } else {
                        acceleration_y = acceleration_y - acceleration * y_proj;
                    }
                }
            }
        }
        // printf("acc_X:%f, acc_Y%f\n", acceleration_x, acceleration_y);
        vx[i] = vx[i] + acceleration_x * dt;
        vy[i] = vy[i] + acceleration_y * dt;
        // printf("v_X:%f, v_Y%f\n\n", vx[i], vy[i]);
    }
}


void generate_data(double *m, double *x,double *y,double *vx,double *vy, int n) {
    // TODO: Generate proper initial position and mass for better visualization
    srand((unsigned)time(NULL));
    for (int i = 0; i < n; i++) {
        m[i] = rand() % max_mass + 1.0f;
        x[i] = 2000.0f + rand() % (bound_x / 4);
        y[i] = 2000.0f + rand() % (bound_y / 4);
        vx[i] = 0.0f;
        vy[i] = 0.0f;
    }
}



void master() {
    double* m = new double[n_body];
    double* x = new double[n_body];
    double* y = new double[n_body];
    double* vx = new double[n_body];
    double* vy = new double[n_body];

    generate_data(m, x, y, vx, vy, n_body);

    Logger l = Logger("cuda", n_body, bound_x, bound_y);

    double *device_m;
    double *device_x;
    double *device_y;
    double *device_vx;
    double *device_vy;

    hipMalloc(&device_m, n_body * sizeof(double));
    hipMalloc(&device_x, n_body * sizeof(double));
    hipMalloc(&device_y, n_body * sizeof(double));
    hipMalloc(&device_vx, n_body * sizeof(double));
    hipMalloc(&device_vy, n_body * sizeof(double));

    hipMemcpy(device_m, m, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_x, x, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_y, y, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_vx, vx, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_vy, vy, n_body * sizeof(double), hipMemcpyHostToDevice);

    int n_block = n_body / block_size + 1;

    for (int i = 0; i < n_iteration; i++){
        std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();

        update_velocity<<<n_block, block_size>>>(device_m, device_x, device_y, device_vx, device_vy, n_body);
        update_position<<<n_block, block_size>>>(device_x, device_y, device_vx, device_vy, n_body);

        hipMemcpy(x, device_x, n_body * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(y, device_y, n_body * sizeof(double), hipMemcpyDeviceToHost);

        std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> time_span = t2 - t1;
        
        printf("Iteration %d, elapsed time: %.3f\n", i, time_span);

        l.save_frame(x, y);

        #ifdef GUI
        glClear(GL_COLOR_BUFFER_BIT);
        glColor3f(1.0f, 0.0f, 0.0f);
        glPointSize(2.0f);
        glBegin(GL_POINTS);
        double xi;
        double yi;
        for (int i = 0; i < n_body; i++){
            xi = x[i];
            yi = y[i];
            glVertex2f(xi, yi);
        }
        glEnd();
        glFlush();
        glutSwapBuffers();
        #else

        #endif

    }

    hipFree(device_m);
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_vx);
    hipFree(device_vy);

    delete[] m;
    delete[] x;
    delete[] y;
    delete[] vx;
    delete[] vy;
    
}


int main(int argc, char *argv[]){
    
    n_body = atoi(argv[1]);
    n_iteration = atoi(argv[2]);

    #ifdef GUI
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
    glutInitWindowPosition(0, 0);
    glutInitWindowSize(500, 500);
    glutCreateWindow("N Body Simulation CUDA Implementation");
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    gluOrtho2D(0, bound_x, 0, bound_y);
    #endif

    master();

    printf("Student ID: 119010001\n"); // replace it with your student id
    printf("Name: Your Name\n"); // replace it with your name
    printf("Assignment 2: N Body Simulation CUDA Implementation\n");

    return 0;

}


